#include "hip/hip_runtime.h"
#include "../HyperGsys/include/dataloader/dataloader.hpp"
#include "../HyperGsys/include/hgnnAgg.cuh"
#include "../HyperGsys/include/spgemm/spgemm.cuh"
#include "../HyperGsys/include/spmm/spmm.cuh"
#include "../HyperGsys/include/util/ramArray.cuh"

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // cusparseSpGEMM
#include <fstream>
#include <stdio.h>  // printf
#include <stdlib.h> // EXIT_FAILURE

__global__ void warm_up() {}

int main(int argc, char **argv) {
  // Host problem definition
  if (argc < 3) {
    printf("Input: first get the path of sparse matrix, then get the "
           "feature length of dense matrix\n");
    exit(1);
  }
  char *filename = argv[1];
  int feature_size = atoi(argv[2]);

  const int iter = 20;
  auto SpPair = DataLoader<Index, DType>(filename);

  std::fstream fs;
  SpMatCsrDescr_t<Index, DType> H = std::get<0>(SpPair);
  SpMatCsrDescr_t<Index, DType> H_T = std::get<1>(SpPair);

  util::RamArray<DType> in_feature(H_T.ncol * feature_size);
  util::RamArray<DType> tmp_feature(H_T.nrow * feature_size);
  util::RamArray<DType> out_feature(H.nrow * feature_size);
  util::RamArray<DType> out_ref(H.nrow * feature_size);

  in_feature.fill_random_h();
  tmp_feature.fill_zero_h();
  out_feature.fill_zero_h();
  in_feature.upload();
  tmp_feature.upload();
  out_feature.upload();
  H.upload();
  H_T.upload();
  printf("start fig8 test\n");
  // warm up
  TwostepSpMM_test<Index, DType, spmm_kernel_met::cusparse>(
      fs, iter, feature_size, H, H_T, in_feature, tmp_feature, out_feature);

  //   if (SpGEMM_SpMM_check<Index, DType>(feature_size, H, H_T, in_feature,
  //                                       tmp_feature, out_ref, out_feature))
  //     SpGEMM_SpMM_test<Index, DType>(fs, iter, feature_size, H, H_T,
  //     in_feature,
  //                                    out_feature);

  HyperGAggr_test<Index, DType, hgnn_kernel_met::edge_based_fused, 2, 32>(
      fs, iter, feature_size, H, H_T, in_feature, out_feature);

  return 0;
}